#include "hip/hip_runtime.h"

#include <stdio.h>

#include <stdlib.h>

//#define BLOCK_SIZE 32
#define SIZE 1024*1024


__host__ void SaveMatrixToFile(char* fileName, int* matrix, int width, int height) {
	FILE* file = fopen(fileName, "wt");
	for (int y = 0; y < height; y++) {
		for (int x = 0; x < width; x++) {
			fprintf(file, "%d\t", matrix[y * width + x]);
		}
		fprintf(file, "\n");
	}
	fclose(file);
}


__global__ void transpose(int* inputMatrix, int* outputMatrix, int width, int height) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	for (int x = 0; x < width; x++)
		for (int y = 0; y < height; y++)
			outputMatrix[x * height + y] = inputMatrix[y * width + x];
			
}

__host__ int main() 
{

	int width;
	int height;
	printf("Input number of columns: ");
	scanf("%d", &width);
	printf("Input number of strings: ");
	scanf("%d", &height);
	int N = width*height;

	hipEvent_t start, stop;
	float gpuTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int* A; 
	A = (int *)malloc(sizeof(int) * N);
	int* A_t;
	A_t = (int *)malloc(sizeof(int) * N);

	for (int i = 0; i < N; i++) 
	{
		A[i] = i + 1;
	}
	SaveMatrixToFile("matrix.txt", A, width, height);


	int* A_dev; 
	int* A_t_dev; 

	hipMalloc((void**)&A_dev, sizeof(int) * N);
	hipMalloc((void**)&A_t_dev, sizeof(int) * N);

	hipMemcpy(A_dev, A, N * sizeof(int), hipMemcpyHostToDevice);

	dim3 block(512);
	hipEventRecord(start, 0);
 
	transpose<<<SIZE/512, block>>>(A_dev, A_t_dev, width, height);
	hipEvent_t syncEvent;
 
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("Time of transposing: %.2f milliseconds\n", gpuTime);  
//	getch();

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(A_t, A_t_dev, N * sizeof(int), hipMemcpyDeviceToHost);
	SaveMatrixToFile("matrix1.txt", A_t, height, width);

	hipFree(A_dev);
	hipFree(A_t_dev);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	delete[] A;
	delete[] A_t;

	return 0;
}
