#include "hip/hip_runtime.h"

#include <stdio.h>


#define SIZE 1024*1024*1000

#define CUDA_CHECK_RETURN(value) {\
        hipError_t _m_cudaStat = value;\
        if (_m_cudaStat != hipSuccess) {\
                fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat),__LINE__, __FILE__);\
                exit(1);\
        }\
}

__host__ int main()
{
	char dev;
	hipSetDevice(dev); 
	hipDeviceProp_t deviceProp; 
	hipGetDeviceProperties(&deviceProp, dev);
	printf("  Total amount of constant memory:  %lu bytes\n", deviceProp.totalConstMem); 
	printf("  Total amount of shared memory per block: %lu bytes\n", deviceProp.sharedMemPerBlock);
	printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock); 
	printf("  Warp size: %d\n", deviceProp.warpSize); 
	printf("  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor); 
	printf("  Maximum number of threads per block:  %d\n", deviceProp.maxThreadsPerBlock);


	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float* vec1 = new float[SIZE];
	hipEventRecord(start, 0);	
	for (int i = 0; i < SIZE; i++)
	{
		vec1[i] = i;
//		printf("#%d\t%f\t %f\n", i, vec1[i]);
	}

        hipEventRecord(stop, 0);
//	float time = 0;

        hipEvent_t syncEvent;
	printf("%g", elapsedTime);
	float* devVec1;
	hipMalloc((void**)&devVec1, sizeof(float) * SIZE);
	hipMemcpy(devVec1, vec1, sizeof(float) * SIZE, hipMemcpyHostToDevice);
	
	hipFree(devVec1);
	delete[] vec1; vec1 = 0;
	return 0;
}
