#include "hip/hip_runtime.h"

#include <stdio.h>


#define CUDA_CHECK_RETURN(value) {\
	hipError_t _m_cudaStat = value;\
	if (_m_cudaStat != hipSuccess) {\
		fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat),__LINE__, __FILE__);\
		exit(1);\
	}\
}

__global__ void addVector(float* left, float* right, float* result)
{
	int idx = threadIdx.x;

	result[idx] = left[idx] + right[idx];
}

#define SIZE 2048 
__host__ int main()
{
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//Âûäåëÿåì ïàìÿòü ïîä âåêòîðà
	float* vec1 = new float[SIZE];
	float* vec2 = new float[SIZE];
	float* vec3 = new float[SIZE];

	for (int i = 0; i < SIZE; i++)
	{
		vec1[i] = i;
		vec2[i] = i;
//		printf("#%d\t%f\t %f\n", i, vec1[i], vec2[i]);
	}

	float* devVec1;
	float* devVec2;
	float* devVec3;

	CUDA_CHECK_RETURN(hipMalloc((void**)&devVec1, sizeof(float) * SIZE));
	CUDA_CHECK_RETURN(hipMalloc((void**)&devVec2, sizeof(float) * SIZE));
	CUDA_CHECK_RETURN(hipMalloc((void**)&devVec3, sizeof(float) * SIZE));

	CUDA_CHECK_RETURN(hipMemcpy(devVec1, vec1, sizeof(float) * SIZE, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(devVec2, vec2, sizeof(float) * SIZE, hipMemcpyHostToDevice));

	
	dim3 block(512);
	hipEventRecord(start,0);
	addVector <<<SIZE/512, block >>>(devVec1, devVec2, devVec3);
	
	hipEventRecord(stop, 0);
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
	hipEvent_t syncEvent;

	CUDA_CHECK_RETURN(hipEventCreate(&syncEvent));
	CUDA_CHECK_RETURN(hipEventRecord(syncEvent, 0));
	CUDA_CHECK_RETURN(hipEventSynchronize(syncEvent));
	CUDA_CHECK_RETURN(hipMemcpy(vec3, devVec3, sizeof(float) * SIZE, hipMemcpyDeviceToHost));
	
	for (int i = 0; i < SIZE; i++)
	{
		//printf("Element #%i: %.1f\n", i, vec3[i]);
	}
	fprintf(stderr,"gTest took %g\n",elapsedTime);

	hipEventDestroy(syncEvent);

	hipFree(devVec1);
	hipFree(devVec2);
	hipFree(devVec3);

	delete[] vec1; vec1 = 0;
	delete[] vec2; vec2 = 0;
	delete[] vec3; vec3 = 0;

	return 0;
}
