
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 10

__global__ void addVector(float* left, float* right, float* result)
{
	int idx = threadIdx.x;
	result[idx] = left[idx] + right[idx];
}

__host__ int main()
{
	float* vec1 = new float[SIZE];
	float* vec2 = new float[SIZE];
	float* vec3 = new float[SIZE];

	for (int i = 0; i < SIZE; i++)
	{
		vec1[i] = i;
		vec2[i] = i;
	}

	float* devVec1;
	float* devVec2;
	float* devVec3;

	hipMalloc((void**)&devVec1, sizeof(float) * SIZE);
	hipMalloc((void**)&devVec2, sizeof(float) * SIZE);
	hipMalloc((void**)&devVec3, sizeof(float) * SIZE);

	hipMemcpy(devVec1, vec1, sizeof(float) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(devVec2, vec2, sizeof(float) * SIZE, hipMemcpyHostToDevice);

	dim3 gridSize = dim3(1, 1, 1);
	dim3 blockSize = dim3(SIZE, 1, 1);

	addVector<<<gridSize, blockSize>>>(devVec1, devVec2, devVec3);
	addVector<<<1, SIZE>>>(devVec1, devVec2, devVec3);

	hipEvent_t syncEvent;

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(vec3, devVec3, sizeof(float) * SIZE, hipMemcpyDeviceToHost);

	for (int i = 0; i < SIZE; i++) 
		printf("Element #%i: %.1f\n", i , vec3[i]);

	hipEventDestroy(syncEvent);

	hipFree(devVec1);
	hipFree(devVec2);
	hipFree(devVec3);

	return EXIT_SUCCESS;
}
