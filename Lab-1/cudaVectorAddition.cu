
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE (1024*1024)

__global__ void addVector(float* left, float* right, float* result)
{
	int idx = threadIdx.x;
	result[idx] = left[idx] + right[idx];
}

__host__ int main()
{
	float* vec1 = new float[SIZE];
	float* vec2 = new float[SIZE];
	float* vec3 = new float[SIZE];

	for (int i = 0; i < SIZE; i++)
	{
		vec1[i] = i;
		vec2[i] = i;
	}

	float* devVec1;
	float* devVec2;
	float* devVec3;

	hipEvent_t start, stop;
	hipMalloc((void**)&devVec1, sizeof(float) * SIZE);
	hipMalloc((void**)&devVec2, sizeof(float) * SIZE);
	hipMalloc((void**)&devVec3, sizeof(float) * SIZE);

	hipMemcpy(devVec1, vec1, sizeof(float) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(devVec2, vec2, sizeof(float) * SIZE, hipMemcpyHostToDevice);

	int block = 512;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	addVector<<<SIZE/512, block>>>(devVec1, devVec2, devVec3);
	hipEventRecord(stop);

	hipEvent_t syncEvent;

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(vec3, devVec3, sizeof(float) * SIZE, hipMemcpyDeviceToHost);

	float time = 0;
//	for (int i = 0; i < SIZE; i++) 
//		printf("Element #%i: %.1f\n", i , vec3[i]);
	hipEventElapsedTime(&time, start, stop);
	printf("Elapsed time: %f\n", time);

	FILE *f = fopen("time.txt", "a+");
	if (f == NULL) {
		fprintf(stderr, "FILE ERROR!\n");
	} else {
		fprintf(f, "%f 512\n", time);
	}
	fclose(f);
	hipEventDestroy(syncEvent);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(devVec1);
	hipFree(devVec2);
	hipFree(devVec3);

	return EXIT_SUCCESS;
}
